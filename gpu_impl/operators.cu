#include "hip/hip_runtime.h"
#include "operators.cuh"

// M1 and M2 are vectors and o is the returned value from Kernel function
__device__ float* Kernel(hipblasHandle_t handle, float* V1, float* V2, int N){
    float* V = (float*)malloc(N * sizeof(float));
    hipMemcpy(V, V1, N * sizeof(float), hipMemcpyDeviceToDevice);
    hipblasSaxpy(handle, N, -1, V2, 1, V, 1);
    float* Out = (float*)malloc(1 * sizeof(float));
    hipblasSnrm2(handle, N, V, 1, Out);
    *Out = 32*expf(-1/0.0006*(*Out));

    free(V);
    return *Out
}
// M1, M2 are square matrix with the same shape and N is the number of rows/columns, Out is the output square matrix of this kernel function with N rows 
__global__ void cov(hipblasHandle_t handle, float* M1, float* M2, int N, float* Out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i=index; i< N; i+=stride){
        for(int j=0; j<N; j++){
            Kernel(handle, M1[i*N], M2[j*N], N, Out[index*N+j]);
        }
    }
}
// M is square matrix and N is the number of rows/columns
__device__ float inv(hipblasHandle_t handle, float* M, int N){
    int *PivotArray = (int*)malloc(N*1*sizeof(int));
    int *infoArray = (int*)malloc(1*sizeof(int));
    float *Carray = (float*)malloc(N*N*sizeof(float));
    hipblasSgetrfBatched(handle, N, &M, N, pivotArray, infoArray, 1);
    hipblasSgetriBatched(handle, N, &M, N, pivotArray, Carray, N, infoArray, 1);

    free(PivotArray);
    free(infoArray);
    return Carray
}

#include "operators.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int N=10;

void print_matrix(float* array, int n){
    for(int i=0; i<n; i++){
        for(int j=0; j<n; j++){
            printf("%.2f, ", array[j*n+i]);
        }
        printf("\n");
    }
}

int main(){

    srand((unsigned int)time(NULL));

    float *M1 = (float*)malloc(N*N*sizeof(float)); 
    float *M2 = (float*)malloc(N*N*sizeof(float));
    float *Out = (float*)malloc(N*N*sizeof(float));
    float a = 5.0;

    // initialize the matrix
    for(int i=0; i<N*N; i++){
        M1[i] = ((float)rand()/(float)(RAND_MAX)) * a;
        M2[i] = ((float)rand()/(float)(RAND_MAX)) * a;
    } 

    // create state variables of CUDA
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // make and copy matrix onto GPU
    float *devM1, *devM2, *devOut;
    int *devN;
    cudaCheckError(hipMalloc((void**)&devM1, N*N*sizeof(float)));
    cudaCheckError(hipMalloc((void**)&devM2, N*N*sizeof(float)));
    cudaCheckError(hipMalloc((void**)&devOut, N*N*sizeof(float)));
    cudaCheckError(hipMalloc((void**)&devN, sizeof(int)));
    cudaCheckError(hipMemcpy(devM1, M1, N * N * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(devM2, M2, N * N * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(devN, &N, sizeof(int), hipMemcpyHostToDevice));

    cov<<<1, N>>>(stat, handle, devM1, devM2, *devN, devOut);
    hipDeviceSynchronize();

    
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
       // print the CUDA error message and exit
       printf("CUDA error: %s\n", hipGetErrorString(error));
       exit(-1);
    }
    

    cudaCheckError(hipMemcpy(Out, devOut, N * N * sizeof(float), hipMemcpyDeviceToHost));
    
    cudaCheckError(hipFree(devM1));
    cudaCheckError(hipFree(devM2));
    cudaCheckError(hipFree(devOut));

    printf("Here is matrix 1:\n");
    print_matrix(M1, N);
    printf("Here is matrix 2:\n");
    print_matrix(M2, N);
    printf("Here is the out:\n");
    print_matrix(Out, N);
    return 0;
}

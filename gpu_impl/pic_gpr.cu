#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <iostream>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <math.h>
#include <string.h>
#include "operators.hpp"

#define NUM_SLAVES 5
#define CARD_SUPPORT_SET 20

// kernel function, called from cov() function (GPU-GPU)
__device__ float Kernel(float *M1, float *M2, int size){
    float *out;
    hipMalloc((void **)&out, size);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSnrm2(handle, size, *M1-*M2, 1, out);
    *out = expf(-1/0.0006*(*out));
    *out = (32*(*out));
    return out;
}

// covariance function (runs on GPU, called by slaves running on GPU)
__global__ void cov(float *A, int size_a, float *B, int size_b, float **out, float (*Kernel)(float *M1, float *M2, int size)) {
    double noise = 0;
    int a = size_a * blockIdx.x;
    int b = size_b * blockIdx.x;

    for(int i = 0; i < size_a; i++){
        float *x = new float[size_b];
        for(int j = 0; j < size_b; j++){
            if(i == j){
                noise = 8.6;
            }else{
                noise = 0;
            }
            x[j] = Kernel(A[a + i], B[b + j], size_b) + noise*noise;
        }
        out[i] = x;
    }
}

// inv function (called from slaves (GPU-GPU))
__device__ float inv(float *a){
    return a;
}

// pseudo-inv function (called from slaves (GPU-GPU))
__device__ float pinv(float *a){
    return a;
}

// to compute local summary (running on GPU)
__global__ void slave_local(float *S, float *D, float *yD, float *U, float *local_M, float *local_C, float (*Kernel)(float *M1, float *M2, int size)) {
    int samples = S.n_rows;
    __shared__ float *SD, *DD, *DS, *SS, *inv_DD_S;

    // host copies
    float *a, *b, **out;

    // device copies
    float *d_a, *d_b, *d_out;

    int s = 4 * sizeof(float*);

    // Allocate space for device copies
    hipMalloc((void **)&d_a, s);
    hipMalloc((void **)&d_b, s);
    hipMalloc((void **)&d_out, s);

    // Calculate for local summary
    // SD = covariance(S, D, Kernel);
    // DD = covariance(D, D, Kernel);
    // DS = covariance(D, S, Kernel);
    // SS = covariance(S, S, Kernel);

    a[0] = S; b[0] = D;
    a[1] = D; b[1] = D;
    a[2] = D; b[2] = S;
    a[3] = S; b[3] = S;

    // copy inputs to device
    hipMemcpy(d_a, &a, s, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, s, hipMemcpyHostToDevice);
    hipMemcpy(d_out, &out, s, hipMemcpyHostToDevice);

    // execute 4 covariance functions in parallel using 4 blocks
    cov<<<4,1>>>(d_a, d_b, d_out, Kernel);

    // synchronice all device functions
    hipDeviceSynchronize();

    // copy outputs to host
    hipMemcpy(out, d_out, s, hipMemcpyDeviceToHost);

    SD = out[0];
    DD = out[1];
    DS = out[2];
    SS = out[3];

    // calculate local summary
    inv_DD_S = inv(DD-DS*inv(SS)*SD);
    local_M = SD*inv_DD_S*yD;
    local_C = SD*inv_DD_S*DS;
}

// to calculate for global summary (running on GPU)
__global__ void slave_global(float *S, float *D, float *yD, float *U, float *local_C, float *global_C, float *global_M, float *pred_mean, float (*Kernel)(float *M1, float *M2, int size)) {
    extern __shared__ float *SD, *DD, *DS, *SS, *inv_DD_S;

    // local copies
    float *a, *b, *out;

    // device copies
    float *d_a, *d_b, *d_out;

    int s = 5 * sizeof(float*);

    // Allocate space for device copies
    hipMalloc((void **)&d_a, s);
    hipMalloc((void **)&d_b, s);
    hipMalloc((void **)&d_out, s);

    // Calculate for global summary
    // mat UU = covariance(U, U, Kernel);
    // mat US = covariance(U, S, Kernel);
    // mat SU = covariance(S, U, Kernel);
    // mat UD = covariance(U, D, Kernel);
    // mat DU = covariance(D, U, Kernel);

    a[0] = U; b[0] = U;
    a[1] = U; b[1] = S;
    a[2] = S; b[2] = U;
    a[3] = U; b[3] = D;
    a[4] = D; b[4] = U;

    // copy inputs to device
    hipMemcpy(d_a, &a, s, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, s, hipMemcpyHostToDevice);
    hipMemcpy(d_out, &out, s, hipMemcpyHostToDevice);

    // execute 5 covariance functions in parallel using 5 blocks
    cov<<<5,1>>>(d_a, d_b, d_out, Kernel);

    // copy outputs to host
    hipMemcpy(out, d_out, s, hipMemcpyDeviceToHost);

    float *UU = out[0];
    float *US = out[1];
    float *SU = out[2];
    float *UD = out[3];
    float *DU = out[4];

    // calculate global summary
    float *local_US = UD*inv_DD_S*DS;
    float *local_SU = SD*inv_DD_S*DU;
    float *local_UU = UD*inv_DD_S*DU;
    float *Phi_US = US+US*pinv(SS)*local_C-local_US;
    float *pred_mean = (Phi_US*pinv(global_C)*global_M) + UD*inv_DD_S*yD;
    float *pred_covar = UU-(Phi_US*pinv(SS)*SU-US*pinv(SS)*local_SU-Phi_US*pinv(global_C)*trans(Phi_US))-local_UU;

    // predictions stored in pred_mean
}

// master runs on CPU
void master(float *S, int** pred, int* partition, float *train_data, float *train_target, float *test_data, float *test_target, int interval, float (*Kernel)(float *M1, float *M2)) {
    int	slaveCount;
    int samples = S.n_rows;
    float *test_mean, *test_covar;

    float *global_M = zeros<mat>(samples, 1);
    float *global_C = covariance(S, S, Kernel);

    float **train_data_arr = new mat [NUM_SLAVES];
    float **train_target_arr = new mat [NUM_SLAVES];
    float **test_data_arr = new mat [NUM_SLAVES];

    float **local_M_arr = new mat [NUM_SLAVES];
    float **local_C_arr = new mat [NUM_SLAVES];

    hipStream_t *streams;
    int s = sizeof(mat);

    // start NUM_SLAVES workers to calculate for local summary
    for (slaveCount = 0; slaveCount < NUM_SLAVES; slaveCount++) {
        // partitions
        train_data_arr[slaveCount] = train_data.rows(slaveCount*interval, (slaveCount+1)*interval-1);
        train_target_arr[slaveCount] = train_target.rows(slaveCount*interval, (slaveCount+1)*interval-1);
        test_data_arr[slaveCount] = test_data.rows(slaveCount*interval, (slaveCount+1)*interval-1);

        // device copies
        mat *d_support, *d_train_data, *d_train_target, *d_test_data, *local_M, *local_C;

        // Allocate space for device copies
        hipMalloc((void **)&d_support, s);
        hipMalloc((void **)&d_train_data, s);
        hipMalloc((void **)&d_train_target, s);
        hipMalloc((void **)&d_test_data, s);

        hipMalloc((void **)&local_M, s);
        hipMalloc((void **)&local_C, s);

        // Copy inputs to device
        hipMemcpy(d_support, &S, s, hipMemcpyHostToDevice);
        hipMemcpy(d_train_data, &train_data_arr[slaveCount], s, hipMemcpyHostToDevice);
        hipMemcpy(d_train_target, &train_target_arr[slaveCount], s, hipMemcpyHostToDevice);
        hipMemcpy(d_test_data, &test_data_arr[slaveCount], s, hipMemcpyHostToDevice);

        // create new stream for parallel grid execution
        hipStreamCreate(&streams[slaveCount]);

        // launch one worker(slave) kernel per stream
        slave_local<<<1, 1, 0, streams[slaveCount]>>>(d_support, d_train_data, d_train_target, d_test_data, local_M, local_C, Kernel);

        // Copy result back to host
        hipMemcpy(&local_M_arr[slaveCount], local_M, s, hipMemcpyDeviceToHost);
        hipMemcpy(&local_C_arr[slaveCount], local_C, s, hipMemcpyDeviceToHost);

        // Cleanup
        hipFree(d_support); hipFree(d_train_data); hipFree(d_train_target); hipFree(d_test_data);
    }

    // synchronice all device functions
    hipDeviceSynchronize();

    // sum up local summary to get global summary
    for (slaveCount = 0; slaveCount < NUM_SLAVES; slaveCount++) {
        global_M = global_M + local_M_arr[slaveCount];
        global_C = global_C + local_C_arr[slaveCount];
    }

    // calculate for final prediction
    for (slaveCount = 0; slaveCount < NUM_SLAVES; slaveCount++) {
        train_data_arr[slaveCount] = train_data.rows(slaveCount*interval, (slaveCount+1)*interval-1);
        train_target_arr[slaveCount] = train_target.rows(slaveCount*interval, (slaveCount+1)*interval-1);
        test_data_arr[slaveCount] = test_data.rows(slaveCount*interval, (slaveCount+1)*interval-1);

        // device copies
        mat *d_support, *d_train_data, *d_train_target, *d_test_data, *local_C;
        mat *d_global_M, *d_global_C;
        double *d_pred_M;

        // Allocate space for device copies
        hipMalloc((void **)&d_support, s);
        hipMalloc((void **)&d_train_data, s);
        hipMalloc((void **)&d_train_target, s);
        hipMalloc((void **)&d_test_data, s);
        hipMalloc((void **)&local_C, s);

        hipMalloc((void **)&d_global_M, s);
        hipMalloc((void **)&d_global_C, s);
        hipMalloc((void **)&d_pred_M, s);

        // Copy inputs to device
        hipMemcpy(d_support, &S, s, hipMemcpyHostToDevice);
        hipMemcpy(d_train_data, &train_data_arr[slaveCount], s, hipMemcpyHostToDevice);
        hipMemcpy(d_train_target, &train_target_arr[slaveCount], s, hipMemcpyHostToDevice);
        hipMemcpy(d_test_data, &test_data_arr[slaveCount], s, hipMemcpyHostToDevice);
        hipMemcpy(local_C, &local_C_arr[slaveCount], s, hipMemcpyHostToDevice);

        hipMemcpy(d_global_M, &global_M, s, hipMemcpyHostToDevice);
        hipMemcpy(d_global_C, &global_C, s, hipMemcpyHostToDevice);

        // launch one worker(slave) kernel per stream, reuse stream to access shared variables
        slave_global<<<1, 1, 0, streams[slaveCount]>>>(d_support, d_train_data, d_train_target, d_test_data, local_C, d_global_M, d_global_C, d_pred_M, Kernel);

        // Copy result back to host
        hipMemcpy(&pred[slaveCount], d_pred_M, sizeof(double), hipMemcpyDeviceToHost);

        // Cleanup
        hipFree(d_support); hipFree(d_train_data); hipFree(d_train_target); hipFree(d_test_data); hipFree(local_C);
        hipFree(d_global_M); hipFree(d_global_C); hipFree(d_pred_M);
    }

    // synchronice all device functions
    hipDeviceSynchronize();

    // results are in pred (int** pred)
    cout<<"Done"<<endl;
}

// main runs on CPU
int main(int argc, char *argv[]){
    // load data from csv file
    std::string path = "data.csv";
    mat data = parseCsvFile(path, 1000);

    // normalise the dataset
    int rows = data.n_rows;
    int columns = data.n_cols;

    mat Max = max(data, 0);
    mat Min = min(data, 0);

    for(int i=0;i<rows;i++){
        // ignore the last target column
        for(int j=1;j<columns; j++){
            data(i,j) = (data(i,j)-Min(0, j))/Max(0, j);
        }
    }

    // split data into training and testing samples
    int all_samples = data.n_rows;
    mat train_data = data.rows(0, all_samples/2-1).cols(1, 8);
    mat train_target = data.rows(0, all_samples/2-1).col(0);
    mat test_data = data.rows(all_samples/2, all_samples-1).cols(1, 8);
    mat test_target = data.rows(all_samples/2, all_samples-1).col(0);

    int *pred = new int[all_samples-all_samples/2];

    // get the support data set and partitions of training data set
    mat support;
    int partitions[NUM_SLAVES+1];
    int intervals = all_samples/(2*NUM_SLAVES);
    for(int i=0;i<NUM_SLAVES;i++){
        partitions[i+1] = all_samples/(2*NUM_SLAVES);
        int idx = i*intervals;
        for(int j=0;j<CARD_SUPPORT_SET/NUM_SLAVES;j++){
            support.insert_rows(0, train_data.row(idx+j));
        }
    }

    // call master function (execute on CPU) to start slaves (working on GPU)
    master(support, &pred, partitions, train_data, train_target, test_data, test_target, intervals, Kernel);

    // print out predictions in pred variable
    mat pred_M = zeros<mat>(all_samples-all_samples/2, 1);
    for(int i = 0; i < (all_samples-all_samples/2); i++){
        cout << pred[i] << "(" << test_target(i, 0) << ")" << "\t";
        if(i%10==0 && i!=0){
            cout<<endl;
        }
        pred_M(i, 0) = pred[i];
    }
    return(0);
}
